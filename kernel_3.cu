#include "hip/hip_runtime.h"
/*
* Copyright (c) 2011-2012, Archaea Software, LLC.
* Copyright (c) 2025, simveit (modifications)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <hip/hip_runtime.h>

#include <hipcub/hipcub.hpp>
#include <iostream>

/*
Kernel 3
*/
template <unsigned int threadsPerBlock, unsigned int batchSize>
__global__ void kernel_3(const int *d_in, int *d_out, size_t N) {
  extern __shared__ int sums[threadsPerBlock];
  int sum = 0;
  const int tid = threadIdx.x;  
  const int global_tid = blockIdx.x * threadsPerBlock + tid;
  const int threads_in_grid = threadsPerBlock * gridDim.x;

  if (global_tid < N) {
#pragma unroll
    for (int j = 0; j < batchSize; j++) {
      if (global_tid * batchSize + j < N) {
        sum += d_in[global_tid * batchSize + j];
      }
    }
  }
  sums[tid] = sum;
  __syncthreads();

#pragma unroll
  for (int activeThreads = threadsPerBlock >> 1; activeThreads > 32;
       activeThreads >>= 1) {
    if (tid < activeThreads) {
      sums[tid] += sums[tid + activeThreads];
    }
    __syncthreads();
  }

  volatile int *volatile_sums = sums;
#pragma unroll
  for (int activeThreads = 32; activeThreads; activeThreads >>= 1) {
    if (tid < activeThreads) {
      volatile_sums[tid] += volatile_sums[tid + activeThreads];
    }
    __syncwarp();
  }

  if (tid == 0) {
    atomicAdd(d_out, volatile_sums[tid]);
  }
}

template <int threadsPerBlock, int batchSize>
void kernel_3_launch(const int *d_in, int *d_out, size_t N) {
  const int numBlocks = (N + threadsPerBlock * batchSize - 1) /
                        (threadsPerBlock * batchSize);
  hipMemset(d_out, 0, sizeof(int));
  kernel_3<threadsPerBlock, batchSize><<<numBlocks, threadsPerBlock>>>(d_in,
                                                                       d_out, N);
}

__global__ void warmupKernel() { extern __shared__ int tmp[]; }

int main() {
  warmupKernel<<<1024, 1024, 1024 * sizeof(int)>>>();
  hipDeviceSynchronize();

  const int N = 1 << 30;
  size_t size = N * sizeof(int);
  const int threadsPerBlock = 512;
  const int batchSize = 12;
  const int numBlocks = (N + threadsPerBlock * batchSize - 1) /
                        (threadsPerBlock * batchSize);

  int *h_in = new int[N];
  int *h_first = new int[numBlocks];
  int h_out = 0.0f;

  srand(42);
  for (int i = 0; i < N; ++i) {
    h_in[i] = rand() % 100;
  }

  int *d_in;
  int *d_first;
  int *d_out;
  hipMalloc(&d_in, size);
  hipMalloc(&d_first, numBlocks * sizeof(int));
  hipMalloc(&d_out, sizeof(int));

  hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);
  hipMemcpy(d_first, h_first, numBlocks * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_out, &h_out, sizeof(int), hipMemcpyHostToDevice);

  kernel_3_launch<threadsPerBlock, batchSize>(d_in, d_out, N);

  hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost);

  int h_out_ref = 0;
  for (int i = 0; i < N; ++i) {
    h_out_ref += h_in[i];
  }
  std::cout << "h_out: " << h_out << ", h_out_ref: " << h_out_ref << std::endl;

  size_t num_runs = 1000;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  for (size_t i = 0; i < num_runs; ++i) {
    kernel_3_launch<threadsPerBlock, batchSize>(d_in, d_out, N);
  }

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  milliseconds /= num_runs;
  std::cout << "Time taken: " << milliseconds << " ms" << std::endl;

  auto bandwidth = N * sizeof(int) / milliseconds / 1e6;
  const auto max_bandwidth = 3.3 * 1e3;  // 3.3 TB/s on H100

  std::cout << "Bandwidth: " << bandwidth << " GB/s" << std::endl;
  std::cout << "% of max bandwidth: " << bandwidth / max_bandwidth * 100
            << std::endl;

  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(d_in);
  hipFree(d_first);
  hipFree(d_out);
  delete[] h_in;
}
