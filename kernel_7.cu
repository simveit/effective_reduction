#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <hipcub/hipcub.hpp>
#include <iostream>

/*
Kernel 7
*/
template <unsigned int threadsPerBlock, unsigned int batchSize>
__global__ void kernel_7(const int *d_in, int *d_out, size_t N) {
  extern __shared__ int sums[threadsPerBlock];
  const int tid = threadIdx.x;  
  const int global_tid = blockIdx.x * threadsPerBlock + tid;
  const int threads_in_grid = threadsPerBlock * gridDim.x;

  int sum = 0;

  if (global_tid < N) {
#pragma unroll
    for (int j = 0; j < batchSize; j++) {
      if (global_tid * batchSize + j < N) {
        sum += d_in[global_tid * batchSize + j];
      }
    }
  }
  sums[tid] = sum;
  __syncthreads();

#pragma unroll
  for (int activeThreads = threadsPerBlock >> 1; activeThreads > 32;
       activeThreads >>= 1) {
    if (tid < activeThreads) {
      sums[tid] += sums[tid + activeThreads];
    }
    __syncthreads();
  }

  int sum_warp;
  
  if (tid < 32) {
    sum_warp = sums[tid] + sums[tid + 32];

    sum_warp = __reduce_add_sync(0xffffffff, sum_warp);
  }

  if (tid == 0) {
    atomicAdd(d_out, sum_warp);
  }
}

template <int threadsPerBlock, int batchSize>
void kernel_7_launch(const int *d_in, int *d_out, size_t N) {
  const int numBlocks = (N + threadsPerBlock * batchSize - 1) /
                        (threadsPerBlock * batchSize);
  hipMemset(d_out, 0, sizeof(int));
  kernel_7<threadsPerBlock, batchSize><<<numBlocks, threadsPerBlock>>>(d_in,
                                                                       d_out, N);
}

__global__ void warmupKernel() { extern __shared__ int tmp[]; }

int main() {
  warmupKernel<<<1024, 1024, 1024 * sizeof(int)>>>();
  hipDeviceSynchronize();

  const int N = 1 << 30;
  size_t size = N * sizeof(int);
  const int threadsPerBlock = 512;
  const int batchSize = 12;
  const int numBlocks = (N + threadsPerBlock * batchSize - 1) /
                        (threadsPerBlock * batchSize);

  int *h_in = new int[N];
  int *h_first = new int[numBlocks];
  int h_out = 0.0f;

  srand(42);
  for (int i = 0; i < N; ++i) {
    h_in[i] = rand() % 100;
  }

  int *d_in;
  int *d_first;
  int *d_out;
  hipMalloc(&d_in, size);
  hipMalloc(&d_first, numBlocks * sizeof(int));
  hipMalloc(&d_out, sizeof(int));

  hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);
  hipMemcpy(d_first, h_first, numBlocks * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_out, &h_out, sizeof(int), hipMemcpyHostToDevice);

  kernel_7_launch<threadsPerBlock, batchSize>(d_in, d_out, N);

  hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost);

  int h_out_ref = 0;
  for (int i = 0; i < N; ++i) {
    h_out_ref += h_in[i];
  }
  std::cout << "h_out: " << h_out << ", h_out_ref: " << h_out_ref << std::endl;

  size_t num_runs = 1000;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  for (size_t i = 0; i < num_runs; ++i) {
    kernel_7_launch<threadsPerBlock, batchSize>(d_in, d_out, N);
  }

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  milliseconds /= num_runs;
  std::cout << "Time taken: " << milliseconds << " ms" << std::endl;

  auto bandwidth = N * sizeof(int) / milliseconds / 1e6;
  const auto max_bandwidth = 3.3 * 1e3;  // 3.3 TB/s on H100

  std::cout << "Bandwidth: " << bandwidth << " GB/s" << std::endl;
  std::cout << "% of max bandwidth: " << bandwidth / max_bandwidth * 100
            << std::endl;

  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(d_in);
  hipFree(d_first);
  hipFree(d_out);
  delete[] h_in;
}