#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <hipcub/hipcub.hpp>
#include <iostream>

/*
Kernel 5
*/
void kernel_5_launch(const int *d_in, int *d_out, size_t N) {
  void* d_temp = nullptr;
  size_t temp_storage = 0;

  // First call to determine temporary storage size
  hipcub::DeviceReduce::Sum(d_temp, temp_storage, d_in, d_out, N);
  
  // Allocate temporary storage
  assert(temp_storage > 0);
  hipMalloc(&d_temp, temp_storage);

  hipcub::DeviceReduce::Sum(d_temp, temp_storage, d_in, d_out, N);
}

__global__ void warmupKernel() { extern __shared__ int tmp[]; }

int main() {
  warmupKernel<<<1024, 1024, 1024 * sizeof(int)>>>();
  hipDeviceSynchronize();

  const int N = 1 << 30;
  size_t size = N * sizeof(int);
  const int threadsPerBlock = 512;
  const int batchSize = 12;
  const int numBlocks = (N + threadsPerBlock * batchSize - 1) /
                        (threadsPerBlock * batchSize);

  int *h_in = new int[N];
  int *h_first = new int[numBlocks];
  int h_out = 0.0f;

  srand(42);
  for (int i = 0; i < N; ++i) {
    h_in[i] = rand() % 100;
  }

  int *d_in;
  int *d_first;
  int *d_out;
  hipMalloc(&d_in, size);
  hipMalloc(&d_first, numBlocks * sizeof(int));
  hipMalloc(&d_out, sizeof(int));

  hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);
  hipMemcpy(d_first, h_first, numBlocks * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_out, &h_out, sizeof(int), hipMemcpyHostToDevice);

  kernel_5_launch(d_in, d_out, N);

  hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost);

  int h_out_ref = 0;
  for (int i = 0; i < N; ++i) {
    h_out_ref += h_in[i];
  }
  std::cout << "h_out: " << h_out << ", h_out_ref: " << h_out_ref << std::endl;

  size_t num_runs = 1000;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  for (size_t i = 0; i < num_runs; ++i) {
    kernel_5_launch(d_in, d_out, N);
  }

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  milliseconds /= num_runs;
  std::cout << "Time taken: " << milliseconds << " ms" << std::endl;

  auto bandwidth = N * sizeof(int) / milliseconds / 1e6;
  const auto max_bandwidth = 3.3 * 1e3;  // 3.3 TB/s on H100

  std::cout << "Bandwidth: " << bandwidth << " GB/s" << std::endl;
  std::cout << "% of max bandwidth: " << bandwidth / max_bandwidth * 100
            << std::endl;

  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(d_in);
  hipFree(d_first);
  hipFree(d_out);
  delete[] h_in;
}