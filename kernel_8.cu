#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <hipcub/hipcub.hpp>
#include <iostream>

/*
Kernel 8
*/
template <unsigned int threadsPerBlock, unsigned int batchSize>
__global__ void kernel_8(const int4 *d_in, int *d_out, size_t N) {
  extern __shared__ int sums[threadsPerBlock];
  int sum = 0;
  const int tid = threadIdx.x;  
  const int global_tid = blockIdx.x * threadsPerBlock + tid;
  const int threads_in_grid = threadsPerBlock * gridDim.x;


  if (global_tid < N) {
#pragma unroll
    for (int i = 0; i < batchSize >> 2; i++) {
      const int4 val = d_in[global_tid * (batchSize >> 2) + i];
      if (global_tid * batchSize + i * 4 < N) {
        sum += val.x + val.y + val.z + val.w;
      }
    }
  }
  sums[tid] = sum;
  __syncthreads();

#pragma unroll
  for (int activeThreads = threadsPerBlock >> 1; activeThreads > 32;
       activeThreads >>= 1) {
    if (tid < activeThreads) {
      sums[tid] += sums[tid + activeThreads];
    }
    __syncthreads();
  }

  int sum_warp;

  if (tid < 32) {
    sum_warp = sums[tid] + sums[tid + 32];
#pragma unroll
    for (int offset = 16; offset > 0; offset >>= 1) {
      sum_warp += __shfl_down_sync(0xffffffff, sum_warp, offset);
    }
  }

  if (tid == 0) {
    atomicAdd(d_out, sum_warp);
  }
}

template <int threadsPerBlock, int batchSize>
void kernel_8_launch(const int *d_in, int *d_out, size_t N) {
  const int numBlocks = (N + threadsPerBlock * batchSize - 1) /
                        (threadsPerBlock * batchSize);
  const int4 *d_in_cast = reinterpret_cast<const int4 *>(d_in);
  hipMemset(d_out, 0, sizeof(int));
  kernel_8<threadsPerBlock, batchSize><<<numBlocks, threadsPerBlock>>>(d_in_cast,
                                                                       d_out, N);
}

__global__ void warmupKernel() { extern __shared__ int tmp[]; }

int main() {
  warmupKernel<<<1024, 1024, 1024 * sizeof(int)>>>();
  hipDeviceSynchronize();

  const int N = 1 << 30;
  size_t size = N * sizeof(int);
  const int threadsPerBlock = 512;
  const int batchSize = 12;
  const int numBlocks = (N + threadsPerBlock * batchSize - 1) /
                        (threadsPerBlock * batchSize);

  int *h_in = new int[N];
  int *h_first = new int[numBlocks];
  int h_out = 0.0f;

  srand(42);
  for (int i = 0; i < N; ++i) {
    h_in[i] = rand() % 100;
  }

  int *d_in;
  int *d_first;
  int *d_out;
  hipMalloc(&d_in, size);
  hipMalloc(&d_first, numBlocks * sizeof(int));
  hipMalloc(&d_out, sizeof(int));

  hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);
  hipMemcpy(d_first, h_first, numBlocks * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_out, &h_out, sizeof(int), hipMemcpyHostToDevice);

  kernel_8_launch<threadsPerBlock, batchSize>(d_in, d_out, N);

  hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost);

  int h_out_ref = 0;
  for (int i = 0; i < N; ++i) {
    h_out_ref += h_in[i];
  }
  std::cout << "h_out: " << h_out << ", h_out_ref: " << h_out_ref << std::endl;
  
  size_t num_runs = 1000;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  for (size_t i = 0; i < num_runs; ++i) {
    kernel_8_launch<threadsPerBlock, batchSize>(d_in, d_out, N);
  }

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  milliseconds /= num_runs;
  std::cout << "Time taken: " << milliseconds << " ms" << std::endl;

  auto bandwidth = N * sizeof(int) / milliseconds / 1e6;
  const auto max_bandwidth = 3.3 * 1e3;  // 3.3 TB/s on H100

  std::cout << "Bandwidth: " << bandwidth << " GB/s" << std::endl;
  std::cout << "% of max bandwidth: " << bandwidth / max_bandwidth * 100
            << std::endl;

  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(d_in);
  hipFree(d_first);
  hipFree(d_out);
  delete[] h_in;
}